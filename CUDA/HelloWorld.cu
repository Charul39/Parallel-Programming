// CUDA -First Programs: “Hello, world” is traditionally the first program we write. We can do the same for CUDA. Here it is:

// In file hello.cu:


#include <hip/hip_runtime.h>
#include "stdio.h"
int main()
{
 printf("Hello, world\n");
 return 0;
}

// On your host machine, you can compile and this with: 

// $ nvcc hello.cu

// Execution on GPU equipped server
// $ ./a.out


// Discussion: 
// You can change the output file name with the –o flag: nvcc –o hello hello.cu
// If you edit your .bashrc file you can also add your current directory to your path if you don’t want to have to type the preceding . all of the time, which refers to the current working directory. 
// Add export PATH=$PATH:.
// To the .bashrc file. Some would recommend not doing this for security purposes.

// The point is that CUDA C programs can do everything a regular C program can do.




// Flow of Program: Open text editor (like vi/vim) open a new file - call it whatever you'd like.  It should do the following:
// •	Use the appropriate .cu include file
// •	compile and this with: $ nvcc hello.cu
// •	Print a hello message that includes its task rank and processor name Execution on       
// •	GPU equipped server$ ./a.out
// •	Terminate the Connection  environment
